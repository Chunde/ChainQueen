#include "hip/hip_runtime.h"
#include "kernels.h"
#include "linalg.h"
#include "state.cuh"
#include <cstdio>
#include <vector>

// For deformation gradient update

// Takes B, dL/dAB
// Returns dL/dA
__device__ Matrix dAB2dA(const Matrix &B, const Matrix &dAB) {
  Matrix dA;
  for (int p = 0; p < dim; p++) {
    for (int q = 0; q < dim; q++) {
      for (int j = 0; j < dim; j++) {
        dA[p][q] += dAB[p][j] * B[q][j];
      }
    }
  }
  return dA;
}

// Takes A, B, dL/dAB
// Returns dL/dB
__device__ Matrix dAB2dB(const Matrix &A, const Matrix &dAB) {
  Matrix dB;
  for (int p = 0; p < dim; p++) {
    for (int q = 0; q < dim; q++) {
      for (int i = 0; i < dim; i++) {
        dB[p][q] += dAB[i][q] * A[i][p];
      }
    }
  }
  return dB;
}

__device__ Vector duTv2du(const Vector &v, const real &duTv) {
  return duTv * v;
}

__device__ Vector duTv2dv(const Vector &u, const real &duTv) {
  return duTv * u;
}

__global__ void P2G_backward(State state, State next_state) {
  // Scatter particle gradients to grid nodes
  // P2G part of back-propagation

  int part_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (part_id >= state.num_particles) {
    return;
  }

  Vector x = state.get_x(part_id), v = state.get_v(part_id);
  Matrix F = state.get_F(part_id);
  Matrix C = state.get_C(part_id);

  auto grad_x_next = next_state.get_grad_x(part_id);
  auto grad_C_next = next_state.get_grad_C(part_id);
  auto grad_v_next = next_state.get_grad_v(part_id);
  auto grad_F_next = next_state.get_grad_F(part_id);
  Matrix G;  // TODO

  // (A) v_p^n+1, accumulate
  grad_v_next = grad_v_next + state.dt * grad_x_next;

  // (B) C_p^n+1, accumulate
  for (int alpha = 0; alpha < dim; alpha++) {
    for (int beta = 0; beta < dim; beta++) {
      for (int gamma = 0; gamma < dim; gamma++) {
        grad_C_next[alpha][beta] +=
            state.dt * grad_F_next[alpha][gamma] * F[beta][gamma];
      }
    }
  }

  next_state.set_grad_v(part_id, grad_v_next);
  next_state.set_grad_C(part_id, grad_C_next);

  TransferCommon<true> tc(state, x);

  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      for (int k = 0; k < dim; k++) {
        real N = tc.w(i, j, k);
        Vector dpos = tc.dpos(i, j, k);

        // (C) v_i^n
        real grad_v_i[dim];
        for (int alpha = 0; alpha < dim; alpha++) {
          grad_v_i[alpha] = grad_v_next[alpha] * N;
          if (mpm_enalbe_apic) {
            for (int beta = 0; beta < dim; beta++) {
              grad_v_i[alpha] += grad_C_next[alpha][beta] * dpos[beta];
            }
          }
        }
        auto grad_n = state.grad_grid_node(
            tc.base_coord[0] + i, tc.base_coord[1] + j, tc.base_coord[2] + k);
        for (int d = 0; d < dim; d++) {
          atomicAdd(&grad_n[d], grad_v_i[d]);
        }
      }
    }
  }
}

__global__ void grid_backward(State state) {
  // Scatter particle gradients to grid nodes
  // P2G part of back-propagation
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < state.num_cells) {
    auto node = state.grid_node(id);
    auto grad_node = state.grad_grid_node(id);
    if (node[dim] > 0) {
      int x = id / (state.res[1] * state.res[2]),
          y = id / state.res[2] % state.res[1], z = id % state.res[2];
      // (D)
      // Convert grad_v to grad_p
      // grad_p = grad_v / m
      auto m = node[dim];
      real inv_m = 1.0f / m;  // TODO: guard?
      auto grad_v_i = state.get_grad_grid_velocity(x, y, z);
      auto grad_p = inv_m * grad_v_i;
      auto v_i = Vector(node);
      auto p_i = m * v_i;
      // (E)
      real grad_m = 0;
      for (int alpha = 0; alpha < dim; alpha++) {
        grad_m -= inv_m * v_i[alpha] * grad_v_i[alpha];
        grad_node[alpha] = grad_p[alpha];
      }
      grad_node[dim] = grad_m;
    }
  }
}

// (F), (G), (H), (I), (J)
__global__ void G2P_backward(State state, State next_state) {
  // Scatter particle gradients to grid nodes
  // P2G part of back-propagation
  int part_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (part_id >= state.num_particles) {
    return;
  }

  auto x = state.get_x(part_id);
  auto v = state.get_v(part_id);
  auto F = state.get_F(part_id);
  auto C = state.get_C(part_id);
  auto P = state.get_P(part_id);

  Matrix grad_P_next = next_state.get_grad_P(part_id);
  Matrix grad_P, grad_F;
  auto grad_F_next = next_state.get_grad_F(part_id);
  auto grad_C_next = next_state.get_grad_C(part_id);
  auto grad_v_next = next_state.get_grad_v(part_id);
  Matrix grad_C;
  auto C_next = next_state.get_C(part_id);

  TransferCommon<true> tc(state, x);
  Vector grad_v;
  real grad_P_scale = state.dt * state.invD * V;

  // (H) term 2
  Times_Rotated_dP_dF_FixedCorotated(mu, lambda, F.data(), grad_P.data(),
                                     grad_F.data());

  for (int alpha = 0; alpha < dim; alpha++) {
    for (int beta = 0; beta < dim; beta++) {
      // (H) term 1
      for (int gamma = 0; gamma < dim; gamma++) {
        grad_F[alpha][beta] +=
            grad_F_next[gamma][beta] *
            (real(gamma == alpha) + state.dt * C_next[gamma][alpha]);
      }
    }
  }

  // (J) term 1
  Vector grad_x = next_state.get_grad_x(part_id);
  // printf("grad_x %f\n", grad_x[0]);
  auto G = state.invD * state.dt * P * transposed(F) + m_p * C;

  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      for (int k = 0; k < dim; k++) {
        real N = tc.w(i, j, k);
        Vector dpos = tc.dpos(i, j, k);
        auto grad_p = state.get_grad_grid_velocity(
            tc.base_coord[0] + i, tc.base_coord[1] + j, tc.base_coord[2] + k);

        auto grad_N = tc.dw(i, j, k);
        real grad_v_i[dim];
        real mi = state.get_grid_mass(
            tc.base_coord[0] + i, tc.base_coord[1] + j, tc.base_coord[2] + k);
        auto vi = state.get_grid_velocity(
            tc.base_coord[0] + i, tc.base_coord[1] + j, tc.base_coord[2] + k);
        real grad_mi =
            state.grad_grid_node(tc.base_coord[0] + i, tc.base_coord[1] + j,
                                 tc.base_coord[2] + k)[dim];
        for (int alpha = 0; alpha < dim; alpha++) {
          // (F) v_p^n
          grad_v[alpha] += N * m_p * grad_p[alpha];
          grad_v_i[alpha] = grad_v_next[alpha] * N;

          /*
          grad_x[alpha] +=
              grad_N[alpha] * (grad_v_next[alpha] * state.invD +
                               grad_p[alpha] * mi * vi[alpha] + m_p * grad_mi);

          // temporally disable
          for (int beta = 0; beta < dim; beta++) {
            grad_x[alpha] += state.invD * grad_C_next[beta][alpha] *
                                 (grad_N[alpha] * vi[alpha] * dpos[beta] -
                                  tc.w(i, j, k) * vi[alpha]) -
                             grad_p[beta] * G[beta][alpha];
          }
          */

          for (int beta = 0; beta < dim; beta++) {
            // (G) P_p^n
            for (int gamma = 0; gamma < dim; gamma++) {
              grad_P[alpha][beta] +=
                  grad_P_scale * grad_p[alpha] * F[gamma][beta] * dpos[gamma];
              // (H), term 3
              grad_F[alpha][beta] +=
                  grad_P_scale * P[gamma][beta] * dpos[alpha];
            }
            grad_v_i[alpha] += grad_C_next[alpha][beta] * dpos[beta];
            // (I) C_p^n
            grad_C[alpha][beta] += grad_p[alpha] * m_p * dpos[beta];
          }
        }
        state.set_grad_grid_velocity(
            i, j, k, Vector(grad_v_i[0], grad_v_i[1], grad_v_i[2]));
      }
    }
  }
  state.set_grad_v(part_id, grad_v);
  state.set_grad_x(part_id, grad_x);
}

void backward(State &state, State &next) {
  state.clear_gradients();
  int num_blocks =
      (state.num_particles + particle_block_dim - 1) / particle_block_dim;
  int num_blocks_grid = state.grid_size();
  P2G_backward<<<num_blocks, particle_block_dim>>>(state, next);
  auto err = hipDeviceSynchronize();
  if (err) {
    printf("Launch: %s\n", hipGetErrorString(err));
    exit(-1);
  }
  grid_backward<<<state.num_cells / grid_block_dim + 1, grid_block_dim>>>(
      state);
  G2P_backward<<<num_blocks, particle_block_dim>>>(state, next);
}

void backward_mpm3d_state(void *state_, void *next_state_) {
  State *state = reinterpret_cast<State *>(state_);
  State *next_state = reinterpret_cast<State *>(next_state_);
  backward(*state, *next_state);
}

void set_grad_loss(void *state_) {
  State *state = reinterpret_cast<State *>(state_);
  state->clear_gradients();
  int num_particles = state->num_particles;
  std::vector<float> grad_x_host(num_particles * dim);
  for (int i = 0; i < num_particles; i++) {
    grad_x_host[i * 3] = 1;
  }
  hipMemcpy(state->grad_x_storage, grad_x_host.data(),
             sizeof(real) * dim * num_particles, hipMemcpyHostToDevice);
}

std::vector<float> fetch_mpm3d_grad_v(void *state_) {
  State *state = reinterpret_cast<State *>(state_);
  return state->fetch_grad_v();
}

std::vector<float> fetch_mpm3d_grad_x(void *state_) {
  State *state = reinterpret_cast<State *>(state_);
  return state->fetch_grad_x();
}
