#include "hip/hip_runtime.h"
#include "kernels.h"
#include "linalg.h"
#include "state.cuh"
#include <cstdio>
#include <vector>

__global__ void saxpy(int n, real a, real *x, real *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

void saxpy_cuda(int N, real alpha, real *x, real *y) {
  real *d_x, *d_y;

  hipMalloc(&d_x, N * sizeof(real));
  hipMalloc(&d_y, N * sizeof(real));

  hipMemcpy(d_x, x, N * sizeof(real), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(real), hipMemcpyHostToDevice);

  saxpy<<<(N + 255) / 256, 256>>>(N, alpha, d_x, d_y);

  hipMemcpy(y, d_y, N * sizeof(real), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
}

// Gather data from SOA
// Ensure coalesced global memory access

// Do not consider sorting for now. Use atomics instead.

inline __device__ Matrix PK1(Matrix F) {
  real J = determinant(F);
  Matrix r, s;
  polar_decomp(F, r, s);
  return (2 * mu * (F - r) * transposed(F) + Matrix(lambda * (J - 1) * J));
}

// One particle per thread
__global__ void P2G(State state) {
  // constexpr int scratch_size = 8;
  //__shared__ real scratch[dim + 1][scratch_size][scratch_size][scratch_size];

  int part_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (part_id >= state.num_particles) {
    return;
  }

  auto inv_dx = state.inv_dx;
  real dt = state.dt;

  Vector x = state.get_x(part_id), v = state.get_v(part_id);
  Matrix F = state.get_F(part_id);
  Matrix C = state.get_C(part_id);

  TransferCommon<> tc(state, x);

  // Fixed corotated
  auto P = PK1(F);
  state.set_P(part_id, P);
  Matrix stress = -4 * inv_dx * inv_dx * dt * V * P;

  auto affine =
      real(mpm_enalbe_force) * stress + real(mpm_enalbe_apic) * m_p * C;

  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      for (int k = 0; k < dim; k++) {
        Vector dpos = tc.dpos(i, j, k);

        real contrib[dim + 1];

        auto tmp = affine * dpos + m_p * v;

        auto w = tc.w(i, j, k);
        contrib[0] = tmp[0] * w;
        contrib[1] = tmp[1] * w;
        contrib[2] = tmp[2] * w;
        contrib[3] = m_p * w;

        auto node = state.grid_node(tc.base_coord[0] + i, tc.base_coord[1] + j,
                                    tc.base_coord[2] + k);
        for (int p = 0; p < dim + 1; p++) {
          atomicAdd(&node[p], contrib[p]);
        }
      }
    }
  }
}

__global__ void G2P(State state, State next_state) {
  int part_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (part_id >= state.num_particles) {
    return;
  }

  auto inv_dx = state.inv_dx;
  real dt = state.dt;

  Vector x = state.get_x(part_id);
  Vector v;
  Matrix F = state.get_F(part_id);
  Matrix C;

  TransferCommon<> tc(state, x);

  for (int i = 0; i < dim; i++) {
    for (int j = 0; j < dim; j++) {
      for (int k = 0; k < dim; k++) {
        Vector dpos = tc.dpos(i, j, k);
        auto node = state.grid_node(tc.base_coord[0] + i, tc.base_coord[1] + j,
                                    tc.base_coord[2] + k);
        auto node_v = Vector(node[0], node[1], node[2]);

        auto w = tc.w(i, j, k);
        v = v + w * node_v;
        C = C + Matrix::outer_product(w * node_v, 4 * inv_dx * inv_dx * dpos);
      }
    }
  }
  next_state.set_x(part_id, x + state.dt * v);
  next_state.set_v(part_id, v);
  next_state.set_F(part_id, (Matrix(1) + dt * C) * F);
  next_state.set_C(part_id, C);
}

__global__ void test_svd(int n, Matrix *A, Matrix *U, Matrix *sig, Matrix *V) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < n) {
    svd(A[id], U[id], sig[id], V[id]);
  }
}

void test_svd_cuda(int n, real *A, real *U, real *sig, real *V) {
  Matrix *d_A, *d_U, *d_sig, *d_V;

  hipMalloc(&d_A, sizeof(Matrix) * (unsigned int)(n));
  hipMemcpy(d_A, A, sizeof(Matrix) * n, hipMemcpyHostToDevice);

  hipMalloc(&d_U, sizeof(Matrix) * (unsigned int)(n));
  hipMalloc(&d_sig, sizeof(Matrix) * (unsigned int)(n));
  hipMalloc(&d_V, sizeof(Matrix) * (unsigned int)(n));

  test_svd<<<(n + 127) / 128, 128>>>(n, d_A, d_U, d_sig, d_V);

  std::vector<Matrix> h_U(n), h_sig(n), h_V(n);
  hipMemcpy(h_U.data(), d_U, sizeof(Matrix) * n, hipMemcpyDeviceToHost);
  hipMemcpy(h_sig.data(), d_sig, sizeof(Matrix) * n, hipMemcpyDeviceToHost);
  hipMemcpy(h_V.data(), d_V, sizeof(Matrix) * n, hipMemcpyDeviceToHost);

  // Taichi uses column-first storage
  for (int p = 0; p < n; p++) {
    for (int i = 0; i < 3; i++) {
      for (int j = 0; j < 3; j++) {
        U[p * 12 + 4 * i + j] = h_U[p][j][i];
        sig[p * 12 + 4 * i + j] = h_sig[p][j][i];
        V[p * 12 + 4 * i + j] = h_V[p][j][i];
      }
    }
  }
}

__global__ void normalize_grid(State state) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int boundary = 3;
  if (id < state.num_cells) {
    auto node = state.grid_node(id);
    if (node[dim] > 0) {
      real inv_m = 1.0f / node[dim];
      node[0] *= inv_m;
      node[1] *= inv_m;
      node[2] *= inv_m;
      for (int i = 0; i < dim; i++) {
        node[i] += state.gravity[i] * state.dt;
      }
      int x = id / (state.res[1] * state.res[2]),
          y = id / state.res[2] % state.res[1], z = id % state.res[2];
      if (x < boundary || y < boundary || y < boundary ||
          x + boundary >= state.res[0] || y + boundary >= state.res[1] ||
          z + boundary >= state.res[2]) {
        // All sticky for now
        /*
        for (int i = 0; i < dim; i++) {
          node[i] = 0;
        }
        */
        node[1] = max(0.0f, node[1]);
      }
    }
  }
}

void advance(State &state, State &new_state) {
  hipMemset(state.grid_storage, 0,
             state.num_cells * (state.dim + 1) * sizeof(real));
  int num_blocks =
      (state.num_particles + particle_block_dim - 1) / particle_block_dim;
  P2G<<<num_blocks, particle_block_dim>>>(state);

  auto err = hipDeviceSynchronize();
  if (err) {
    printf("Launch: %s\n", hipGetErrorString(err));
    exit(-1);
  }
  normalize_grid<<<(state.grid_size() + grid_block_dim - 1) / grid_block_dim,
                   grid_block_dim>>>(state);
  G2P<<<num_blocks, particle_block_dim>>>(state, new_state);
}

void MPMKernelLauncher(
    int res[dim], int num_particles, real dx, real dt, real gravity[dim],
    const real *inx, const real *inv, const real *inF, const real *inC,
    real *outx, real *outv, real *outF, real *outC,
    real *outP, real *outgrid) {
  printf("MPM Kernel Launch~~\n");
  auto instate = new State(res, num_particles, dx, dt, gravity, 
      (real *)inx, (real *)inv, (real *)inF, (real *)inC, outP, outgrid);
  auto outstate = new State(res, num_particles, dx, dt, gravity, 
      outx, outv, outF, outC, NULL, NULL);
  advance(*instate, *outstate);
  printf("MPM Kernel Finish~~\n");
}

void initialize_mpm3d_state(int *res,
                            int num_particles,
                            float *gravity,
                            void *&state_,
                            float dt,
                            float *initial_positions) {
  // State(int res[dim], int num_particles, real dx, real dt, real
  auto state = new State(res, num_particles, 1.0f / res[0], dt, gravity);
  state_ = state;
  hipMemcpy(state->x_storage, initial_positions,
             sizeof(Vector) * num_particles, hipMemcpyHostToDevice);
}

void forward_mpm3d_state(void *state_, void *new_state_) {
  State *state = reinterpret_cast<State *>(state_);
  State *new_state = reinterpret_cast<State *>(new_state_);
  advance(*state, *new_state);
}

std::vector<float> fetch_mpm3d_particles(void *state_) {
  State *state = reinterpret_cast<State *>(state_);
  return state->fetch_x();
}

void set_initial_velocities(void *state_, float *v) {
  State *state = reinterpret_cast<State *>(state_);
  hipMemcpy(state->v_storage, v, sizeof(real) * dim * state->num_particles,
             hipMemcpyHostToDevice);
}

void set_initial_F(void *state_, float *F) {
  State *state = reinterpret_cast<State *>(state_);
  hipMemcpy(state->F_storage, F,
             sizeof(real) * dim * dim * state->num_particles,
             hipMemcpyHostToDevice);
}
