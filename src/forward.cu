#include "hip/hip_runtime.h"
#include "kernels.h"
#include "linalg.cuh"
#include <cstdio>
#include "../../../../../../../usr/local/cuda/include/hip/driver_types.h"

void run(real *a, real *b, real *c) {

}

__global__ void saxpy_g(int n, real a, real *x, real *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

void saxpy_cuda(int N, real alpha, real *x, real *y) {
  real* d_x, *d_y;
  /*
  hipMalloc(&d_x, n * sizeof(real));
  hipMalloc(&d_y, n * sizeof(real));
  hipMemcpy(d_x, x, n * sizeof(real), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(real), hipMemcpyHostToDevice);
  for (int i = 0; i < n; i++) {
    printf("%f %f\n", x[i], y[i]);
  }
  printf("size %d\n", n * sizeof(real));
  saxpy_g<<<1, 256>>>(n, alpha, x, y);
  hipMemcpy(y, d_y, n * sizeof(real), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) {
    printf("%f %f\n", x[i], y[i]);
  }
  hipFree(d_x);
  hipFree(d_y);
  printf("done\n");
  */

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  saxpy_g<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-i * 4));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
}

void test() {
  int N = 256;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  saxpy_g<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
