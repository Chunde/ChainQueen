#include "hip/hip_runtime.h"
#include "kernels.h"
#include "linalg.cuh"
#include <cstdio>
#include "../../../../../../../usr/local/cuda/include/hip/driver_types.h"

void run(real *a, real *b, real *c) {

}

__global__ void saxpy_g(int n, real a, real *x, real *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  x[i] = 0;
  y[i] = 0;
  //if (i < n) {
    //y[i] = a * x[i] + y[i];
  //}
}

void saxpy_cuda(int n, real alpha, real *x, real *y) {
  real* d_x, *d_y;
  hipMalloc(&d_x, n * sizeof(real));
  hipMalloc(&d_y, n * sizeof(real));
  hipMemcpy(d_x, x, n * sizeof(real), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(real), hipMemcpyHostToDevice);
  for (int i = 0; i < n; i++) {
    printf("%f %f\n", x[i], y[i]);
  }
  printf("size %d\n", n * sizeof(real));
  saxpy_g<<<1, 256>>>(n, alpha, x, y);
  hipMemcpy(y, d_y, n * sizeof(real), hipMemcpyDeviceToHost);
  hipMemcpy(x, d_x, n * sizeof(real), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < n; i++) {
    printf("%f %f\n", x[i], y[i]);
  }
  hipFree(d_x);
  hipFree(d_y);
  printf("done\n");
}
